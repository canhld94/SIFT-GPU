#include "hip/hip_runtime.h"
#include "sift.hpp"
#include "sift_cuda.hpp"
#include "cuda_helper.h"

#define MAX_THREADS_PER_BLOCK 1024

__global__ void gaussianBlurRow(float* input_g, float* output, float* filter, int rows, int cols, int filter_size){
    extern __shared__ float shared_ptr[];
    float* filter_s = shared_ptr;
    float* input_s = (float*)&shared_ptr[filter_size];

    int half_size = filter_size/2;
    int row_idx = blockIdx.x;
    int col_idx = blockIdx.y * blockDim.x + threadIdx.x;
    int input_idx = row_idx * cols + col_idx;

    // Filter to shared memory
    if(threadIdx.x < filter_size){
        filter_s[threadIdx.x] = filter[threadIdx.x];
    }
    if(blockDim.x < filter_size && threadIdx.x == 0){
        for(int j = blockDim.x ; j < filter_size ; j++){
            filter_s[j] = filter[j];
        }
    }

    // Input image to shared memory
    int row_offset = row_idx * cols - half_size;
    if(threadIdx.x < cols + half_size*2){
        if(threadIdx.x < half_size || threadIdx.x >= cols + half_size){
            input_s[threadIdx.x] = 0;
        }else{
            input_s[threadIdx.x] = input_g[row_offset + threadIdx.x];
        }
    }
    if(blockDim.x < cols + half_size*2 && threadIdx.x == 0){
        for(int j = blockDim.x ; j < cols + half_size*2 ; j++){
            if(j < half_size || j >= cols + half_size){
                input_s[j] = 0;
            }else{
                input_s[j] = input_g[row_offset + j];
            }
        }
    }
    __syncthreads();
    if(col_idx >= cols)
        return;
    float tmp = 0;
    for(int i = -half_size ; i < half_size ; i++){
        tmp += input_s[col_idx + i + half_size] * filter_s[i + half_size];
    }
    output[input_idx] = tmp;
}
__global__ void gaussianBlurCol(float* input_g, float* output, float* filter, int rows, int cols, int filter_size){
    extern __shared__ float shared_ptr[];
    float* filter_s = shared_ptr;
    float* input_s = (float*)&shared_ptr[filter_size];

    int half_size = filter_size/2;
    int row_idx = blockIdx.y * blockDim.x + threadIdx.x;
    int col_idx = blockIdx.x;
    int input_idx = row_idx * cols + col_idx;

    // Filter to shared memory
    if(threadIdx.x < filter_size){
        filter_s[threadIdx.x] = filter[threadIdx.x];
    }
    if(blockDim.x < filter_size && threadIdx.x == 0){
        for(int j = blockDim.x ; j < filter_size ; j++){
            filter_s[j] = filter[j];
        }
    }

    // Input image to shared memory
    if(threadIdx.x < rows + half_size*2){
        if(threadIdx.x < half_size || threadIdx.x >= rows + half_size){
            input_s[threadIdx.x] = 0;
        }else{
            input_s[threadIdx.x] = input_g[(threadIdx.x - half_size)*cols + col_idx];
        }
    }
    if(blockDim.x < rows + half_size*2 && threadIdx.x == 0){
        for(int j = blockDim.x ; j < rows + half_size*2 ; j++){
            if(j < half_size || j >= rows + half_size){
                input_s[j] = 0;
            }else{
                input_s[j] = input_g[(j - half_size)*cols + col_idx];
            }
        }
    }
    __syncthreads();
    if(row_idx >= rows)
        return;
    float tmp = 0;
    for(int i = -half_size ; i < half_size ; i++){
        tmp += input_s[row_idx + i + half_size] * filter_s[i + half_size];
    }
    output[input_idx] = tmp;
}

__global__ void gaussianFilter1D(float* kernel_data,float sigma,int w){
    int tid = threadIdx.x;
    double dat = 1. / sqrt(2 * PI * sigma * sigma) * exp(-((double) (tid - w) * (tid - w)) * 1. / (2 * sigma * sigma));
    kernel_data[tid] = (float)dat;
}

__global__ void halfImage(float* input, float* output, int rows, int cols){
    int tid_row = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_col = blockIdx.y * blockDim.y + threadIdx.y;

    int dst_cols = cols/2;

    if( tid_col * 2 > cols || tid_row * 2 > rows)
        return;

    output[ tid_row * dst_cols + tid_col ] = input[ (tid_row*2) * cols + (tid_col*2) ];
}

__global__ void differentiate(float* input1, float* input2, float* output, int rows, int cols){
    int tid_row = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_col = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = tid_row * cols + tid_col;
    if(tid >= rows*cols)
        return;
    output[tid] = input2[tid] - input1[tid];
}

extern "C" void getGaussianKernel1DGPU(float sigma, float* d_kernel_data, int w){
    int size = 2 * w + 1;
    dim3 block(size);
    gaussianFilter1D<<<1,block>>>(d_kernel_data,sigma,w);
    CHECK(hipGetLastError());
    CHECK(hipDeviceSynchronize());
}

extern "C" void gaussianBlur1DGPU(int octave_idx, int scale_idx, float* src, float* dst, float* dst_h, float* filter, float* itm_data, int rows, int cols, int filter_size, hipStream_t stream) {
    //CHECK(hipMalloc((float**)&itm_data, rows*cols * sizeof(float)));

    dim3 gridRow(rows,cols/MAX_THREADS_PER_BLOCK + (cols%MAX_THREADS_PER_BLOCK?1:0));
    dim3 blockRow(MIN(cols,MAX_THREADS_PER_BLOCK));
    gaussianBlurRow<<<gridRow,blockRow,( filter_size + (cols + (filter_size/2)*2) )*sizeof(float),stream>>>(src,itm_data,filter,rows,cols,filter_size);
    CHECK(hipGetLastError());
    //CHECK(hipDeviceSynchronize());
    CHECK(hipStreamSynchronize(stream));

    dim3 gridCol(cols,rows/MAX_THREADS_PER_BLOCK + (rows%MAX_THREADS_PER_BLOCK?1:0));
    dim3 blockCol(MIN(rows,MAX_THREADS_PER_BLOCK));
    gaussianBlurCol<<<gridCol,blockCol,( filter_size + (rows + (filter_size/2)*2) )*sizeof(float),stream>>>(itm_data,dst,filter,rows,cols,filter_size);
    CHECK(hipGetLastError());
    //CHECK(hipDeviceSynchronize());
    //CHECK(hipFree(itm_data));
    CHECK(hipStreamSynchronize(stream));
    CHECK(hipMemcpyAsync(dst_h,dst, rows * cols * sizeof(float),hipMemcpyDeviceToHost,stream));
}

extern "C" void halfImageGPU(float* src, float* dst, float* dst_h, int rows_ori, int cols_ori, hipStream_t stream){
    int rows = rows_ori / 2;
    int cols = cols_ori / 2;
    dim3 grid(rows/32 + (rows%32?1:0),cols/32 + (cols%32?1:0));
    dim3 block(MIN(rows,32),MIN(cols,32));
    halfImage<<<grid,block,0,stream>>>(src,dst,rows_ori,cols_ori);
    CHECK(hipGetLastError());
    //CHECK(hipDeviceSynchronize());
    CHECK(hipStreamSynchronize(stream));
    CHECK(hipMemcpyAsync(dst_h,dst, rows * cols * sizeof(float),hipMemcpyDeviceToHost,stream));
}

extern "C" void differentiateGPU(float* src1, float* src2, float* dst, float* dst_h, int rows, int cols, hipStream_t stream){
    dim3 grid(rows/32 + (rows%32?1:0),cols/32 + (cols%32?1:0));
    dim3 block(MIN(rows,32),MIN(cols,32));
    differentiate<<<grid,block,0,stream>>>(src1, src2, dst, rows, cols);
    CHECK(hipGetLastError());
    //CHECK(hipDeviceSynchronize());
    CHECK(hipStreamSynchronize(stream));
    CHECK(hipMemcpyAsync(dst_h,dst, rows * cols * sizeof(float),hipMemcpyDeviceToHost,stream));
}

extern "C" void SIFT_NCL_GPU(InputArray image,
    std::vector<KeyPoint> & keypoints,
    OutputArray descriptors){
        std::vector<Mat> dogpyr(nOctaves*(nScales - 1));
        std::vector<Mat> gpyr(nOctaves*nScales);
        float* dummy;
        CHECK(hipMalloc((float**)&dummy,1*sizeof(float)));
        CHECK(hipFree(dummy));
        double t, tf;
        tf = getTickFrequency();
        t = (double) getTickCount();
        float** h_gpyr_arr = new float*[nOctaves*nScales];
        float** h_dogpyr_arr = new float*[nOctaves*(nScales-1)];
        float** gpyr_arr = new float*[nOctaves*nScales];
        float** dogpyr_arr = new float*[nOctaves*(nScales-1)];
        float** filter_arr = new float*[nScales];
        float** itm_arr = new float*[nScales];
        std::vector<int> filter_sizes(nScales);
        std::vector<int> rows(nOctaves);
        std::vector<int> cols(nOctaves);
        std::vector<hipStream_t> streams(nScales);
        Mat img = image.getMat();
        prepareForGPU(img, filter_arr, gpyr_arr, h_gpyr_arr, gpyr, dogpyr_arr, h_dogpyr_arr, dogpyr, itm_arr, rows, cols, filter_sizes, streams);
        for(int i = 0 ; i < nOctaves ; i++){
            buildGaussianPyramidGPU(i, img, filter_arr, gpyr_arr, gpyr, itm_arr, rows, cols, filter_sizes,streams);
            buildDoGPyramidGPU(i, gpyr_arr, dogpyr_arr, dogpyr, rows, cols,streams);
        }
        prepareForCPU(gpyr_arr, dogpyr_arr, gpyr, dogpyr, itm_arr, rows, cols, streams);
        t = (double) getTickCount() - t;
        printf("pyramid construction time: %g\n", t*1000./tf);
        //char s[100];
	    //for (int i = 0; i < gpyr.size(); ++i) {
	    //    sprintf(s, "DoGaussian %d.png", i);
	    //    normalize(gpyr[i], gpyr[i], 0, 255, NORM_MINMAX);
	    //    imwrite(s, gpyr[i]);
	    //}
        t = (double) getTickCount();
        findScaleSpaceExtrema(gpyr,dogpyr,keypoints, 5);
        t = (double) getTickCount() - t;
        printf("keypoint localization time: %g\n", t*1000./tf);
        t = (double) getTickCount();
        int dsize = SIFT_DESCR_WIDTH*SIFT_DESCR_WIDTH*SIFT_DESCR_HIST_BINS;
        descriptors.create((int)keypoints.size(), dsize, CV_32F);
        Mat _descriptors = descriptors.getMat();
        calDescriptor(gpyr, keypoints, _descriptors, 0);
        t = (double) getTickCount() - t;
        printf("descriptor extraction time: %g\n", t*1000./tf);
        CHECK(hipDeviceReset());
        return;
}

extern "C" void prepareForGPU(Mat img_mat, float** filter, float** gpyr, float** h_gpyr,std::vector<Mat>& gpyr_mat, float** dogpyr, float** h_dogpyr, std::vector<Mat>& dogpyr_mat, float** itm_data, std::vector<int>& rows, std::vector<int>& cols, std::vector<int>& filter_sizes, std::vector<hipStream_t>& streams){
	std::vector<float> sig(nScales);
    float *img_arr = (float *) img_mat.data;
    int row_ori = img_mat.rows;
    int col_ori = img_mat.cols;

    double k = pow(2.0, 1.0/nOctaveLayers);
    for(int i = 0 ; i < nOctaves ; i++){
        rows[i] = row_ori;
        cols[i] = col_ori;
        row_ori = floor(row_ori/2);
        col_ori = floor(col_ori/2);
        
        for(int j = 0 ; j < nScales ; j++){
            CHECK(hipMalloc((float**)&(gpyr[i*nScales + j]), rows[i] * cols[i] * sizeof(float) ));
            CHECK(hipHostMalloc((float**)&(h_gpyr[i*nScales + j]), rows[i] * cols[i] * sizeof(float)));
            gpyr_mat[i*nScales + j] = cv::Mat(rows[i], cols[i], CV_32F, h_gpyr[i*nScales + j], cols[i]*sizeof(float));
        }
        for(int j = 0 ; j < nScales - 1 ; j++){
            CHECK(hipMalloc((float**)&(dogpyr[i*(nScales - 1) + j]), rows[i] * cols[i] * sizeof(float) ));
            CHECK(hipHostMalloc((float**)&(h_dogpyr[i*(nScales-1) + j]), rows[i] * cols[i] * sizeof(float)));
            dogpyr_mat[i*(nScales-1) + j] = cv::Mat(rows[i], cols[i], CV_32F, h_dogpyr[i*(nScales-1) + j],cols[i]*sizeof(float));
        }
    }
	for(int i = 0; i < nScales; i++){
        hipStreamCreate(&(streams[i]));
        if(i == 0){
            sig[i] = Sigma;
        }else{
            double sig_total = pow(k* 1.0, (double) i)*Sigma;
            sig[i] = (float) sqrt(sig_total*sig_total - Sigma*Sigma);
        }

        int w = floor(3 * sig[i]);
        int size = 2 * w + 1;
        filter_sizes[i] = size;

        CHECK(hipMalloc((float**)&itm_data[i],rows[0]*cols[0]*sizeof(float)));
        CHECK(hipMalloc((float**)&filter[i], size * sizeof(float)));
        getGaussianKernel1DGPU(sig[i],filter[i],w);
	}
    CHECK(hipMemcpy(gpyr[0],img_arr,rows[0] * cols[0] * sizeof(float),hipMemcpyHostToDevice));
    
    double init_sigma = sqrt(Sigma*Sigma + 0.5*0.5);
    float* init_kernel;
    int init_w = floor(3 * init_sigma);
    int init_ksize = 2 * init_w + 1;

    CHECK(hipMalloc((float**)&init_kernel, init_ksize * sizeof(float)));
    getGaussianKernel1DGPU(init_sigma, init_kernel, init_w);
    gaussianBlur1DGPU(0,0,gpyr[0],gpyr[0],(float*)gpyr_mat[0].data,init_kernel,itm_data[0],rows[0],cols[0],init_ksize,streams[0]);
}

extern "C" void prepareForCPU(float** gpyr_d, float** dogpyr_d, std::vector<Mat>& gpyr, std::vector<Mat>& dogpyr, float** itm_data, std::vector<int>& rows, std::vector<int>& cols, std::vector<hipStream_t>& streams){
    CHECK(hipDeviceSynchronize());
}

extern "C" void buildGaussianPyramidGPU(int octave_idx, Mat image, float** filter, float** gpyr, std::vector<Mat>& gpyr_mat , float** itm_data, std::vector<int>& rows, std::vector<int>& cols, std::vector<int>& filter_sizes, std::vector<hipStream_t>& streams){
    for(int j = 0 ; j < nScales ; j++ ){
        if(octave_idx == 0 && j == 0){
        }else if(j == 0){
            halfImageGPU(gpyr[(octave_idx-1)*nScales + nOctaveLayers],gpyr[octave_idx*nScales],(float*)gpyr_mat[octave_idx*nScales+j].data,rows[octave_idx-1],cols[octave_idx-1],streams[j]);
        }else{
            gaussianBlur1DGPU(octave_idx,j,gpyr[octave_idx*nScales],gpyr[octave_idx*nScales+j],(float*)gpyr_mat[octave_idx*nScales+j].data,filter[j],itm_data[j],rows[octave_idx],cols[octave_idx],filter_sizes[j],streams[j]);
        }
    }
}
extern "C" void buildDoGPyramidGPU(int octave_idx, float** gpyr, float** dogpyr, std::vector<Mat>& dogpyr_mat, std::vector<int>& rows, std::vector<int>& cols, std::vector<hipStream_t>& streams){
    for(int j = 0 ; j < nScales - 1 ; j++ ){
        differentiateGPU(gpyr[octave_idx*nScales+j],gpyr[octave_idx*nScales+j+1],dogpyr[octave_idx*(nScales - 1) + j],(float*)dogpyr_mat[octave_idx*(nScales-1)+j].data, rows[octave_idx], cols[octave_idx],streams[j]);
    }
}
